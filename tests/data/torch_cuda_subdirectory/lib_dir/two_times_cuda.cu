#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/ops/zeros_like.h>
#include <c10/cuda/CUDAException.h>

#ifndef __CUDACC_EXTENDED_LAMBDA__
    #error "Modified CUDA_NVCC_FLAGS (extended lambda) from torch not correctly propagated"
#endif

template <class T>
__global__ void
two_times_kernel(const T* const input, T* const output, const std::size_t N)
{
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    {
        output[i] = T(2) * input[i];
    }
}

at::Tensor
two_times(const at::Tensor& input)
{
    auto output = at::zeros_like(input);

    AT_DISPATCH_ALL_TYPES(input.scalar_type(),
                          "two_times_cuda",
                          [&]()
                          {
                              const std::uint32_t block_size = 128;
                              const std::uint32_t num_blocks = (input.numel() + block_size - 1) / block_size;
                              two_times_kernel<<<num_blocks, block_size>>>(input.data_ptr<scalar_t>(),
                                                                           output.data_ptr<scalar_t>(),
                                                                           input.numel());
                              C10_CUDA_KERNEL_LAUNCH_CHECK();
                          });

    return output;
}
